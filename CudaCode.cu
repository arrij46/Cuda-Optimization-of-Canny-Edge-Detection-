#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define BLOCK 16
#define DIM 16
#define VERBOSE 0
#define BOOSTBLURFACTOR 90.0

extern "C" void launchKernel2(short int *smoothedim, int rows, int cols, short int **delta_x, short int **delta_y);
extern "C" void launchKernel(int center, unsigned char *image, float *kernel, float *tempim, int rows, int cols, short int **smoothedim, int *windowsize);

__global__ void GaussianBlurX(int center, unsigned char *image, float *kernel, float *tempim, int rows, int cols)
{
    // printf("Hello from CUDA Kernel!\n");
    float dot, /* Dot product summing variable. */
        sum;   /* Sum of the kernel weights variable. */

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (r >= rows || c >= cols)
        return;

    /****************************************************************************
     * Blur in the x - direction.
     ****************************************************************************/
    if (VERBOSE)
        printf("   Bluring the image in the X-direction.\n");

    dot = 0.0;
    sum = 0.0;
    for (int cc = (-center); cc <= center; cc++)
    {
        if (((c + cc) >= 0) && ((c + cc) < cols))
        {
            dot += (float)image[r * cols + (c + cc)] * kernel[center + cc];
            sum += kernel[center + cc];
        }
    }
    tempim[r * cols + c] = dot / sum;
}
__global__ void GaussianBlurY(int center, float *kernel, float *tempim, int rows, int cols, short int *smoothedim)
{
    // printf("Hello from CUDA Kernel!\n");
    float dot, /* Dot product summing variable. */
        sum;   /* Sum of the kernel weights variable. */

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (r >= rows || c >= cols)
        return;
    /****************************************************************************
     * Blur in the y - direction.
     ****************************************************************************/
    if (VERBOSE)
        printf("   Bluring the image in the Y-direction.\n");

    sum = 0.0;
    dot = 0.0;
    for (int rr = (-center); rr <= center; rr++)
    {
        if (((r + rr) >= 0) && ((r + rr) < rows))
        {
            dot += tempim[(r + rr) * cols + c] * kernel[center + rr];
            sum += kernel[center + rr];
        }
    }
    smoothedim[r * cols + c] = (short int)(dot * BOOSTBLURFACTOR / sum + 0.5);
}

__global__ void derivative_x_y_L1(short int *smoothedim, int rows, int cols, short int *delta_x)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < rows && c < cols)
    {
        if (c > 0 && c < cols - 1)
        {
            delta_x[r * cols + c] = smoothedim[r * cols + (c + 1)] - smoothedim[r * cols + (c - 1)];
        }
        else if (c == 0)
        {
            delta_x[r * cols + c] = smoothedim[r * cols + (c + 1)] - smoothedim[r * cols + c];
        }
        else if (c == cols - 1)
        {
            delta_x[r * cols + c] = smoothedim[r * cols + c] - smoothedim[r * cols + (c - 1)];
        }
    }

    // for(r=0;r<rows;r++)

    /*{
        pos = r * cols;
        delta_x[pos] = smoothedim[pos + 1] - smoothedim[pos];
        pos++;
        for (c = 1; c < (cols - 1); c++, pos++)
        {
            delta_x[pos] = smoothedim[pos + 1] - smoothedim[pos - 1];
        }
        delta_x[pos] = smoothedim[pos] - smoothedim[pos - 1];
    }
    */
}
__global__ void derivative_x_y_L2(short int *smoothedim, int rows, int cols, short int *delta_y)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < rows && c < cols)
    {
        if (r > 0 && r < rows - 1)
        {
            delta_y[r * cols + c] = smoothedim[(r + 1) * cols + c] - smoothedim[(r - 1) * cols + c];
        }
        else if (r == 0)
        {
            delta_y[r * cols + c] = smoothedim[(r + 1) * cols + c] - smoothedim[r * cols + c];
        }
        else if (r == rows - 1)
        {
            delta_y[r * cols + c] = smoothedim[r * cols + c] - smoothedim[(r - 1) * cols + c];
        }
    }
}

void launchKernel(int center, unsigned char *image, float *kernel, float *tempim, int rows, int cols, short int **smoothedim, int *windowsize)
{
    int size = rows * cols;
    unsigned char *d_image;
    float *d_kernel;
    float *d_tempim;
    short int *d_smoothedim;
    hipError_t err;
    float milliseconds = 0;

    // Allocate device memory
    hipEvent_t memAllocStart, memAllocStop;
    hipEventCreate(&memAllocStart);
    hipEventCreate(&memAllocStop);
    hipEventRecord(memAllocStart);

    hipMalloc((void **)&d_image, size * sizeof(char));
    hipMalloc((void **)&d_tempim, size * sizeof(float));
    hipMalloc((void **)&d_kernel, (*windowsize) * sizeof(float));
    hipMalloc((void **)&d_smoothedim, size * sizeof(short int));

    hipEventRecord(memAllocStop);
    hipEventSynchronize(memAllocStop);
    hipEventElapsedTime(&milliseconds, memAllocStart, memAllocStop);
    printf("Memory Allocation Time: %.3f ms\n", milliseconds);

    // Copy data to GPU
    hipEvent_t memCopyStart, memCopyStop;
    hipEventCreate(&memCopyStart);
    hipEventCreate(&memCopyStop);
    hipEventRecord(memCopyStart);

    hipMemcpy(d_image, image, size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_tempim, tempim, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, (*windowsize) * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(memCopyStop);
    hipEventSynchronize(memCopyStop);
    hipEventElapsedTime(&milliseconds, memCopyStart, memCopyStop);
    printf("Memory Copy (Host to Device) Time: %.3f ms\n", milliseconds);

    int c = (cols + BLOCK - 1) / BLOCK;
    int r = (rows + BLOCK - 1) / BLOCK;
    dim3 numofblocks(c, r);
    dim3 threadperBlock(BLOCK, BLOCK);

    // Measure Kernel Execution Time
    hipEvent_t kernelStart, kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);
    hipEventRecord(kernelStart);

    printf("Without Shared memory\n");

    GaussianBlurX<<<numofblocks, threadperBlock>>>(center, d_image, d_kernel, d_tempim, rows, cols);
    hipDeviceSynchronize();

    GaussianBlurY<<<numofblocks, threadperBlock>>>(center, d_kernel, d_tempim, rows, cols, d_smoothedim);
    hipDeviceSynchronize();

    hipEventRecord(kernelStop);
    hipEventSynchronize(kernelStop);
    hipEventElapsedTime(&milliseconds, kernelStart, kernelStop);
    printf("Kernel Execution Time: %.3f ms\n", milliseconds);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Kernel launch error: %s\n", hipGetErrorString(err));
        return;
    }

    // Copy data to GPU
    hipEvent_t memCopyBackStart, memCopyBackStop;
    hipEventCreate(&memCopyBackStart);
    hipEventCreate(&memCopyBackStop);
    hipEventRecord(memCopyBackStart);

    hipMemcpy(*smoothedim, d_smoothedim, size * sizeof(short int), hipMemcpyDeviceToHost);

    hipEventRecord(memCopyBackStop);
    hipEventSynchronize(memCopyBackStop);
    hipEventElapsedTime(&milliseconds, memCopyBackStart, memCopyBackStop);
    printf("Memory Copy (Device to Host) Time: %.3f ms\n", milliseconds);

    // Cuda Free

    hipFree(d_image);
    hipFree(d_tempim);
    hipFree(d_kernel);
    hipFree(d_smoothedim);

    hipEventDestroy(memAllocStart);
    hipEventDestroy(memAllocStop);
    hipEventDestroy(memCopyStart);
    hipEventDestroy(memCopyStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    hipEventDestroy(memCopyBackStart);
    hipEventDestroy(memCopyBackStop);
}

void launchKernel2(short int *smoothedim, int rows, int cols, short int **delta_x, short int **delta_y)
{
    int size = rows * cols;
    short int *d_delta_x, *d_delta_y, *d_smoothedim;
    float milliseconds = 0;

    // Allocate device memory
    hipEvent_t memAllocStart, memAllocStop;
    hipEventCreate(&memAllocStart);
    hipEventCreate(&memAllocStop);
    hipEventRecord(memAllocStart);

    hipMalloc((void **)&d_delta_y, size * sizeof(short int));
    hipMalloc((void **)&d_delta_x, size * sizeof(short int));
    hipMalloc((void **)&d_smoothedim, size * sizeof(short int));

    hipEventRecord(memAllocStop);
    hipEventSynchronize(memAllocStop);
    hipEventElapsedTime(&milliseconds, memAllocStart, memAllocStop);
    printf("Memory Allocation Time: %.3f ms\n", milliseconds);

    // Copy data to GPU
    hipEvent_t memCopyStart, memCopyStop;
    hipEventCreate(&memCopyStart);
    hipEventCreate(&memCopyStop);
    hipEventRecord(memCopyStart);

    hipMemcpy(d_smoothedim, smoothedim, size * sizeof(short int), hipMemcpyHostToDevice);

    hipEventRecord(memCopyStop);
    hipEventSynchronize(memCopyStop);
    hipEventElapsedTime(&milliseconds, memCopyStart, memCopyStop);
    printf("Memory Copy (Host to Device) Time: %.3f ms\n", milliseconds);

    int c = (cols + BLOCK - 1) / BLOCK;
    int r = (rows + BLOCK - 1) / BLOCK;
    dim3 numofblocks(c, r);
    dim3 threadperBlock(BLOCK, BLOCK);

    // Measure Kernel Execution Time
    hipEvent_t kernelStart, kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);
    hipEventRecord(kernelStart);

    printf("Without Shared memory 2a\n");

    derivative_x_y_L1<<<numofblocks, threadperBlock>>>(d_smoothedim, rows, cols, d_delta_x);
    hipDeviceSynchronize();

    derivative_x_y_L2<<<numofblocks, threadperBlock>>>(d_smoothedim, rows, cols, d_delta_y);
    hipDeviceSynchronize();

    hipEventRecord(kernelStop);
    hipEventSynchronize(kernelStop);
    hipEventElapsedTime(&milliseconds, kernelStart, kernelStop);
    printf("Kernel Execution Time: %.3f ms\n", milliseconds);

    // Copy data from GPU to CPU
    hipEvent_t memCopyBackStart, memCopyBackStop;
    hipEventCreate(&memCopyBackStart);
    hipEventCreate(&memCopyBackStop);
    hipEventRecord(memCopyBackStart);

    hipMemcpy(*delta_x, d_delta_x, size * sizeof(short int), hipMemcpyDeviceToHost);
    hipMemcpy(*delta_y, d_delta_y, size * sizeof(short int), hipMemcpyDeviceToHost);

    hipEventRecord(memCopyBackStop);
    hipEventSynchronize(memCopyBackStop);
    hipEventElapsedTime(&milliseconds, memCopyBackStart, memCopyBackStop);
    printf("Memory Copy (Device to Host) Time: %.3f ms\n", milliseconds);

    // Cuda Free
    hipEventDestroy(memAllocStart);
    hipEventDestroy(memAllocStop);
    hipEventDestroy(memCopyStart);
    hipEventDestroy(memCopyStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    hipEventDestroy(memCopyBackStart);
    hipEventDestroy(memCopyBackStop);

    hipFree(d_delta_x);
    hipFree(d_delta_y);
    hipFree(d_smoothedim);
}
